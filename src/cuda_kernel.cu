#include "hip/hip_runtime.h"
#include <vector>
#include <cuda_xxhash64.cuh>
#include <graph.h>

typedef unsigned long long int uint64_cu;
typedef uint64_cu vec_t_cu;

class CudaSketch {
  public:
    vec_t* d_bucket_a;
    vec_hash_t* d_bucket_c;

    vec_t failure_factor; 
    size_t num_elems;
    size_t num_buckets;
    size_t num_guesses;
    uint64_t seed;

    // Default Constructor of CudaSketch
    CudaSketch():d_bucket_a(nullptr), d_bucket_c(nullptr) {};

    CudaSketch(vec_t* d_bucket_a, vec_hash_t* d_bucket_c, vec_t failure_factor, size_t num_elems, size_t num_buckets, size_t num_guesses, uint64_t seed): 
      d_bucket_a(d_bucket_a), d_bucket_c(d_bucket_c), failure_factor(failure_factor), num_elems(num_elems), num_buckets(num_buckets), num_guesses(num_guesses), seed(seed) {};
};

__device__ col_hash_t bucket_col_index_hash(const vec_t_cu& update_idx, const long seed_and_col) {
  return CUDA_XXH64(&update_idx, sizeof(update_idx), seed_and_col);
}

__device__ vec_hash_t bucket_index_hash(const vec_t_cu& index, long sketch_seed) {
  return CUDA_XXH32(&index, sizeof(index), sketch_seed);
}

__device__  bool bucket_contains(const vec_t_cu& col_index_hash, const vec_t_cu& guess_nonzero) {
  return (col_index_hash & guess_nonzero) == 0; // use guess_nonzero (power of 2) to check ith bit
}

__device__ void bucket_update(vec_t_cu& a, vec_hash_t& c, const vec_t_cu& update_idx, const vec_hash_t& update_hash) {
  atomicXor(&a, update_idx);
  atomicXor(&c, update_hash);
}

__device__ void bucket_a_single_update(vec_t_cu& a, const vec_t_cu& update_idx) {
  atomicXor(&a, update_idx);
}

__device__ void bucket_c_single_update(vec_hash_t& c, const vec_hash_t& update_hash) {
  atomicXor(&c, update_hash);
}

// Kernel code for only sketch updates
// Old version, needs to be updated
__global__ void sketch_update(int num_updates, vec_t* update_indexes, CudaSketch* cudaSketches) {

  // Get thread id
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // One thread will be responsible for one update to sketch
  if(tid < num_updates) {
    // Step 1: Get cudaSketch
    CudaSketch curr_cudaSketch = cudaSketches[0];

    // Step 2: Get update_idx
    vec_t update_idx = update_indexes[tid];

    // Step 3: Get all the buckets from cudaSketch
    vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
    vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;
    size_t num_elems = curr_cudaSketch.num_elems;

    // Step 4: Get update_hash
    vec_hash_t update_hash = bucket_index_hash(update_idx, curr_cudaSketch.seed);

    bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], update_idx, update_hash);

    // Step 5: Update current sketch
    for (unsigned i = 0; i < curr_cudaSketch.num_buckets; ++i) {
      col_hash_t col_index_hash = bucket_col_index_hash(update_idx, curr_cudaSketch.seed + i);
      for (unsigned j = 0; j < curr_cudaSketch.num_guesses; ++j) {
        unsigned bucket_id = i * curr_cudaSketch.num_guesses + j;
        if (bucket_contains(col_index_hash, ((col_hash_t)1) << j)){
          bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], update_idx, update_hash);
        } else break;
      }
    }
  }
}

// Kernel code of handling all the stream updates
// Four threads will be responsible for one edge update -> two threads are modifying one node's sketches
// Split the workload for traversing through a node's sketch
/*__global__ void quadStream_update(int* nodeUpdates, int num_updates, int num_nodes, int num_sketches, vec_t* edgeUpdates, 
                                    CudaSketch* cudaSketches) {

  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < num_updates * 4){

    // Step 1: Get node based on tid.
    const vec_t_cu node = nodeUpdates[tid / 2];

    // Step 2: Update node's sketches
    for (int i = 0; i < num_sketches; i++) {

      CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + i];
      size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[tid / 2], curr_cudaSketch.seed);

      // Let only one thread to update
      if (tid % 2 == 0) {
        bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[tid / 2], update_hash);        
      }

      __syncthreads();

      // Update node's sketches, splitting workloads for two threads
      for (unsigned j = tid % 2; j < curr_cudaSketch.num_buckets; j += 2) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[tid / 2], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[tid / 2], update_hash);
          } else break;
        }
      }
    }
  }
}*/

// Kernel code of handling all the stream updates
// Four threads will be responsible for one edge update -> two threads are modifying one node's sketches
// One thread is reponsible for bucket_a, the other thread is responsble for bucket_c
__global__ void quadStream_update(int* nodeUpdates, int num_updates, int num_nodes, int num_sketches, vec_t* edgeUpdates, 
                                    CudaSketch* cudaSketches) {

  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < num_updates * 4){

    if (tid < num_updates * 2) { // Current thread is reponsible for bucket_a
      // Step 1: Get node based on tid.
      const vec_t_cu node = nodeUpdates[tid];

      // Step 2: Update node's sketches
      for (int i = 0; i < num_sketches; i++) {

        CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + i];
        size_t num_elems = curr_cudaSketch.num_elems;

        // Get buckets based on current sketch and node id
        vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;

        bucket_a_single_update(bucket_a[num_elems - 1], edgeUpdates[tid]);        
        
        // Update node's sketches, splitting workloads for two threads
        for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
          col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[tid], curr_cudaSketch.seed + j);
          for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
            unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
            if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
              bucket_a_single_update(bucket_a[bucket_id], edgeUpdates[tid]);
            } else break;
          }
        }
      }
    }
    else { // Current thread is reponsible for bucket_c
      // Step 1: Get node based on tid.
      const vec_t_cu node = nodeUpdates[tid - (num_updates * 2)];

      // Step 2: Update node's sketches
      for (int i = 0; i < num_sketches; i++) {

        CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + i];
        size_t num_elems = curr_cudaSketch.num_elems;

        // Get buckets based on current sketch and node id
        vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

        vec_hash_t update_hash = bucket_index_hash(edgeUpdates[tid - (num_updates * 2)], curr_cudaSketch.seed);

        bucket_c_single_update(bucket_c[num_elems - 1], update_hash);   
        
        // Update node's sketches, splitting workloads for two threads
        for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
          col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[tid - (num_updates * 2)], curr_cudaSketch.seed + j);
          for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
            unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
            if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
              bucket_c_single_update(bucket_c[bucket_id], update_hash);
            } else break;
          }
        }
      }
    }
  }
}

// Kernel code of handling all the stream updates
// Two threads will be responsible for one edge update -> one thread is only modifying one node's sketches.
/*__global__ void doubleStream_update(int* nodeUpdates, int num_updates, int num_nodes, int num_sketches, vec_t* edgeUpdates, 
                                    CudaSketch* cudaSketches) {

  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < num_updates * 2){

    // Step 1: Get node based on tid.
    const vec_t_cu node = nodeUpdates[tid];

    // Step 2: Update node's sketches
    for (int i = 0; i < num_sketches; i++) {

      CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + i];
      size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[tid], curr_cudaSketch.seed);

      bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[tid], update_hash);

      // Update node's sketches
      for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[tid], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[tid], update_hash);
          } else break;
        }
      }
    }
  }
}*/

// Kernel code of handling all the stream updates
// Two threads will be responsible for one edge update -> one thread is only modifying one node's sketches.
__global__ void doubleStream_update(int* nodeUpdates, int num_updates, int num_nodes, int num_sketches, vec_t* edgeUpdates, 
                                    CudaSketch* cudaSketches) {

  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < num_updates * 2){

    int offset = tid % num_sketches;
    int start_index = (tid / num_sketches) * num_sketches;

    // Step 2: Update node's sketches
    for (int i = 0; i < num_sketches; i++) {
      // To prevent going out of bounds
      if (start_index + i >= num_updates * 2) {
        break;
      }

      // Step 1: Get node based on tid.
      const vec_t_cu node = nodeUpdates[start_index + i];
      CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + offset];

      size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[start_index + i], curr_cudaSketch.seed);

      bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[start_index + i], update_hash);

      // Update node's sketches
      for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[start_index + i], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[start_index + i], update_hash);
          } else break;
        }
      }
    }
  }
}

// Kernel code of handling all the stream updates
/*__global__ void singleStream_update(int* nodeUpdates, int num_updates, int num_nodes, int num_sketches, vec_t* edgeUpdates, 
                                    CudaSketch* cudaSketches) {

  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // One thread will be responsible for one edge update = one thread is updating sketches on each endpoint nodes (2).
  if (tid < num_updates){
    // Step 1: Get two endpoint nodes based on tid.
    int node1_id = tid * 2;
    int node2_id = (tid * 2) + 1;

    const vec_t_cu node1 = nodeUpdates[node1_id];
    const vec_t_cu node2 = nodeUpdates[node2_id];

    // Step 2a: Update node1's sketches
    for (int i = 0; i < num_sketches; i++) {

      CudaSketch curr_cudaSketch = cudaSketches[(node1 * num_sketches) + i];
      const size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[node1_id], curr_cudaSketch.seed);

      bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[node1_id], update_hash);

      // Update node1's sketches
      for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[node1_id], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[node1_id], update_hash);
          } else break;
        }
      }
    }

    // Step 2b: Update node2's sketches
    for (int i = 0; i < num_sketches; i++) {

      CudaSketch curr_cudaSketch = cudaSketches[(node2 * num_sketches) + i];
      size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[node2_id], curr_cudaSketch.seed);

      bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[node2_id], update_hash);
      
      // Update node2's sketches
      for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[node2_id], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[node2_id], update_hash);
          } else break;
        }
      }
    }
  }
}*/

// Kernel code of handling all the stream updates
__global__ void singleStream_update(int* nodeUpdates, int num_updates, int num_nodes, int num_sketches, vec_t* edgeUpdates, 
                                    CudaSketch* cudaSketches) {

  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // One thread will be responsible for one edge update = one thread is updating sketches on each endpoint nodes (2).
  if (tid < num_updates){
    // Step 1: Get two endpoint nodes based on tid.
    int node1_id = tid * 2;
    int node2_id = (tid * 2) + 1;

    int offset = node1_id % num_sketches;
    int start_index = (node1_id / num_sketches) * num_sketches;

    // Step 2a: Update node1's sketches
    for (int i = 0; i < num_sketches; i++) {
      // To prevent going out of bounds
      if (start_index + i >= num_updates * 2) {
        break;
      }

      const vec_t_cu node = nodeUpdates[start_index + i];
      CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + offset];

      const size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[start_index + i], curr_cudaSketch.seed);

      bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[start_index + i], update_hash);

      // Update node1's sketches
      for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[start_index + i], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[start_index + i], update_hash);
          } else break;
        }
      }
    }

    offset = node2_id % num_sketches;
    start_index = (node2_id / num_sketches) * num_sketches;

    // Step 2b: Update node2's sketches
    for (int i = 0; i < num_sketches; i++) {
      // To prevent going out of bounds
      if (start_index + i >= num_updates * 2) {
        break;
      }

      const vec_t_cu node = nodeUpdates[start_index + i];
      CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + offset];

      size_t num_elems = curr_cudaSketch.num_elems;

      // Get buckets based on current sketch and node id
      vec_t_cu* bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* bucket_c = curr_cudaSketch.d_bucket_c;

      vec_hash_t update_hash = bucket_index_hash(edgeUpdates[start_index + i], curr_cudaSketch.seed);

      bucket_update(bucket_a[num_elems - 1], bucket_c[num_elems - 1], edgeUpdates[start_index + i], update_hash);
      
      // Update node2's sketches
      for (unsigned j = 0; j < curr_cudaSketch.num_buckets; ++j) {
        col_hash_t col_index_hash = bucket_col_index_hash(edgeUpdates[start_index + i], curr_cudaSketch.seed + j);
        for (unsigned k = 0; k < curr_cudaSketch.num_guesses; ++k) {
          unsigned bucket_id = j * curr_cudaSketch.num_guesses + k;
          if (bucket_contains(col_index_hash, ((col_hash_t)1) << k)){
            bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edgeUpdates[start_index + i], update_hash);
          } else break;
        }
      }
    }
  }
}

// Function that calls sketch update kernel code.
void sketchUpdate(int num_threads, int num_blocks, int num_updates, vec_t* update_indexes, CudaSketch* cudaSketches) {
  // Call kernel code
  sketch_update<<<num_blocks, num_threads>>>(num_updates, update_indexes, cudaSketches);
  hipDeviceSynchronize();
}


// Function that calls stream update kernel code.
void streamUpdate(int num_threads, int num_blocks, int *nodeUpdates, size_t num_updates, node_id_t num_nodes, int num_sketches, int num_elems, vec_t *edgeUpdates, 
                  CudaSketch* cudaSketches, int num_threads_per_update) {

  if(num_threads_per_update == 1) { // Updating sketches with one thread per edge update
    singleStream_update<<<num_blocks, num_threads>>>(nodeUpdates, num_updates, num_nodes, num_sketches, edgeUpdates, cudaSketches);
    hipDeviceSynchronize();
  }
  else if(num_threads_per_update == 2) { // Updating sketches with two thread per edge update
    doubleStream_update<<<num_blocks, num_threads>>>(nodeUpdates, num_updates, num_nodes, num_sketches, edgeUpdates, cudaSketches);
    hipDeviceSynchronize();
  }
  else if(num_threads_per_update == 4) { // Updating sketches with four threads per edge update
    quadStream_update<<<num_blocks, num_threads>>>(nodeUpdates, num_updates, num_nodes, num_sketches, edgeUpdates, cudaSketches);
    hipDeviceSynchronize();
  }
  else {
    std::cout << "(cuda_kernel.cu) ERROR: Invalid number of threads per edge update. Must be 1, 2, or 4." << std::endl;
    return;
  }
}
