#include <vector>
#include <graph.h>
#include <graph_worker.h>
#include <map>
#include <binary_graph_stream.h>
#include <cuda_graph.cuh>

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: stream_file, graph_workers, reader_threads" << std::endl;
    exit(EXIT_FAILURE);
  }

  std::string stream_file = argv[1];
  int num_threads = std::atoi(argv[2]);
  if (num_threads < 1) {
    std::cout << "ERROR: Invalid number of graph workers! Must be > 0." << std::endl;
    exit(EXIT_FAILURE);
  }
  int reader_threads = std::atoi(argv[3]);

  BinaryGraphStream_MT stream(stream_file, 1024*32);
  node_id_t num_nodes = stream.nodes();
  size_t num_updates  = stream.edges();
  std::cout << "Running process_stream with CUDA: " << std::endl;
  std::cout << "Processing stream: " << stream_file << std::endl;
  std::cout << "nodes       = " << num_nodes << std::endl;
  std::cout << "num_updates = " << num_updates << std::endl;
  std::cout << std::endl;

  CudaGraph cudaGraph;

  auto config = GraphConfiguration().gutter_sys(CACHETREE).num_groups(num_threads);
  // Configuration is from cache_exp.cpp
  config.gutter_conf().page_factor(1)
              .buffer_exp(20)
              .fanout(64)
              .queue_factor(8)
              .num_flushers(2)
              .gutter_factor(1)
              .wq_batch_per_elm(8);
  Graph g{num_nodes, config, &cudaGraph, reader_threads};

  Supernode** supernodes;
  supernodes = g.getSupernodes();

  // Get variable from sample supernode
  int num_sketches = supernodes[0]->get_num_sktch();
  
  // Get variables from sample sketch
  size_t num_elems = supernodes[0]->get_sketch(0)->get_num_elems();
  size_t num_columns = supernodes[0]->get_sketch(0)->get_columns();
  size_t num_guesses = supernodes[0]->get_sketch(0)->get_num_guesses();

  std::cout << "num_sketches: " << num_sketches << "\n";
  std::cout << "num_elems: " << num_elems << "\n";
  std::cout << "num_columns: " << num_columns << "\n";
  std::cout << "num_guesses: " << num_guesses << "\n";

  // Start timer for initializing
  auto init_start = std::chrono::steady_clock::now();

  GutteringSystem *gts = g.getGTS();
  int batch_size = gts->gutter_size() / sizeof(node_id_t);
  int stream_multiplier = 4;

  std::cout << "Batch_size: " << batch_size << "\n";
  
  CudaUpdateParams* cudaUpdateParams;
  gpuErrchk(hipMallocManaged(&cudaUpdateParams, sizeof(CudaUpdateParams)));
  cudaUpdateParams[0] = CudaUpdateParams(num_nodes, num_updates, num_sketches, num_elems, num_columns, num_guesses, num_threads, batch_size, stream_multiplier);

  CudaSketch* cudaSketches;
  gpuErrchk(hipMallocManaged(&cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch)));

  long* sketchSeeds;
  gpuErrchk(hipMallocManaged(&sketchSeeds, num_nodes * num_sketches * sizeof(long)));

  // Allocate space for all buckets
  vec_t* d_bucket_a;
  vec_hash_t* d_bucket_c;
  gpuErrchk(hipMallocManaged(&d_bucket_a, (num_nodes * num_sketches * num_elems * sizeof(vec_t))));
  gpuErrchk(hipMallocManaged(&d_bucket_c, (num_nodes * num_sketches * num_elems * sizeof(vec_hash_t))));

  for (size_t i = 0; i < (num_nodes * num_sketches * num_elems); i++) {
    d_bucket_a[i] = 0;
    d_bucket_c[i] = 0;
  }

  // Create a vector of cuda supernodes and sketches
  for (int i = 0; i < num_nodes; i++) {
    for (int j = 0; j < num_sketches; j++) {
      Sketch* sketch = supernodes[i]->get_sketch(j);

      int bucket_id = (i * num_sketches * num_elems) + (j * num_elems);
      vec_t* bucket_a = &d_bucket_a[bucket_id];
      vec_hash_t* bucket_c = &d_bucket_c[bucket_id];

      // Rewrite sketch's bucket_a and bucket_c memory location
      sketch->set_bucket_a(bucket_a);
      sketch->set_bucket_c(bucket_c);

      CudaSketch cudaSketch(bucket_a, bucket_c, sketch->get_seed());
      cudaSketches[(i * num_sketches) + j] = cudaSketch;
      sketchSeeds[(i * num_sketches) + j] = sketch->get_seed();
    }
  }

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";

  int maxBytes = num_elems * num_sketches * sizeof(vec_t_cu) + num_elems * num_sketches * sizeof(vec_hash_t);
  cudaGraph.cudaKernel.kernelUpdateSharedMemory(maxBytes);
  std::cout << "Allocated Shared Memory of: " << maxBytes << "\n";

  // Prefetch memory to device 
  gpuErrchk(hipMemPrefetchAsync(cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch), device_id));
  gpuErrchk(hipMemPrefetchAsync(sketchSeeds, num_nodes * num_sketches * sizeof(long), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_a, num_nodes * num_sketches * num_elems * sizeof(vec_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_c, num_nodes * num_sketches * num_elems * sizeof(vec_hash_t), device_id));

  cudaGraph.configure(cudaUpdateParams, cudaSketches, sketchSeeds, num_threads);
  
  MT_StreamReader reader(stream);
  GraphUpdate upd;

  std::cout << "Finished initializing CUDA parameters\n";
  std::chrono::duration<double> init_time = std::chrono::steady_clock::now() - init_start;
  std::cout << "CUDA parameters init duration: " << init_time.count() << std::endl;

  // Start timer for kernel
  auto ins_start = std::chrono::steady_clock::now();

  // Call kernel code
  std::cout << "Update Kernel Starting...\n";

  // Do the edge updates
  std::vector<std::thread> threads;
  threads.reserve(reader_threads);
  auto task = [&](const int thr_id) {
    MT_StreamReader reader(stream);
    GraphUpdate upd;
    while(true) {
      upd = reader.get_edge();
      if (upd.type == BREAKPOINT) break;
      Edge &edge = upd.edge;

      gts->insert({edge.src, edge.dst}, thr_id);
      std::swap(edge.src, edge.dst);
      gts->insert({edge.src, edge.dst}, thr_id);
    }
  };

  // start inserters
  for (int t = 0; t < reader_threads; t++) {
    threads.emplace_back(task, t);
  }
  // wait for inserters to be done
  for (int t = 0; t < reader_threads; t++) {
    threads[t].join();
  }

  auto flush_start = std::chrono::steady_clock::now();
  gts->force_flush();
  GraphWorker::pause_workers();
  hipDeviceSynchronize();
  auto flush_end = std::chrono::steady_clock::now();

  std::cout << "Update Kernel finished.\n";

  // End timer for kernel
  auto ins_end = std::chrono::steady_clock::now();

  /*for (int i = 0; i < cudaGraph.loop_times.size(); i++) {
    std::cout << "Stream #" << i << ": ";
    double total_loop_time = 0;
    for (int j = 0; j < cudaGraph.loop_times[i].size(); j++) {
      total_loop_time += cudaGraph.loop_times[i][j];
    }
    std::cout << total_loop_time << "\n";
  }*/
  
  // Update graph's num_updates value
  g.num_updates += num_updates * 2;

  // Start timer for cc
  auto cc_start = std::chrono::steady_clock::now();
  auto CC_num = g.connected_components().size();

  std::chrono::duration<double> insert_time = flush_end - ins_start;
  std::chrono::duration<double> cc_time = std::chrono::steady_clock::now() - cc_start;
  std::chrono::duration<double> flush_time = flush_end - flush_start;
  std::chrono::duration<double> cc_alg_time = g.cc_alg_end - g.cc_alg_start;

  double num_seconds = insert_time.count();
  std::cout << "Total insertion time(sec):    " << num_seconds << std::endl;
  std::cout << "Updates per second:           " << stream.edges() / num_seconds << std::endl;
  std::cout << "Total CC query latency:       " << cc_time.count() << std::endl;
  std::cout << "  Flush Gutters(sec):           " << flush_time.count() << std::endl;
  std::cout << "  Boruvka's Algorithm(sec):     " << cc_alg_time.count() << std::endl;
  std::cout << "Connected Components:         " << CC_num << std::endl;
}
