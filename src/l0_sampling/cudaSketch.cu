#include "hip/hip_runtime.h"
#include "../../include/l0_sampling/cudaSketch.cuh"

#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include <iterator>

// __global__ means this is called from the CPU, and runs on the GPU
__global__ void sketch_query(const vec_t* bucket_a, const vec_hash_t* bucket_c, vec_t* result, size_t num_guesses, uint64_t seed) {
    int bucketId = blockIdx.y * blockDim.y + threadIdx.y;
    int guessId = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned bucket_id = bucketId * num_guesses + guessId;

    if (bucket_is_good(bucket_a[bucket_id], bucket_c[bucket_id], bucketId, 1 << guessId, seed)) {
      result[0] = bucket_a[bucket_id];
    }
}

__host__ __device__  col_hash_t bucket_col_index_hash(const vec_t& update_idx, const long seed_and_col) {
  return col_hash(&update_idx, sizeof(update_idx), seed_and_col);
}

__host__ __device__  vec_hash_t bucket_index_hash(const vec_t& index, long sketch_seed) {
  return vec_hash(&index, sizeof(index), sketch_seed);
}

__host__ __device__  bool bucket_contains(const col_hash_t& col_index_hash, const col_hash_t& guess_nonzero) {
  return (col_index_hash & guess_nonzero) == 0; // use guess_nonzero (power of 2) to check ith bit
}

__host__ __device__ bool bucket_is_good(const vec_t& a, const vec_hash_t& c, const unsigned bucket_col, const vec_t& guess_nonzero, const long& sketch_seed) {
  return c == bucket_index_hash(a, sketch_seed)
    && bucket_contains(bucket_col_index_hash(a, sketch_seed + bucket_col), guess_nonzero);
}

CudaSketch::CudaSketch(size_t numElements, size_t numBuckets, size_t numGuesses, vec_t* bucketA, vec_hash_t* bucketC, uint64_t currentSeed) {
  num_elements = numElements;
  num_buckets = numBuckets;
  num_guesses = numGuesses;
  bucket_a = bucketA;
  bucket_c = bucketC;
  seed = currentSeed;
  result[0] = 0;
}

void CudaSketch::query() {
  vec_t bucket_a_bytes = sizeof(vec_t) * num_elements;
  vec_hash_t bucket_c_bytes = sizeof(vec_hash_t) * num_elements;
  vec_t result_bytes = sizeof(vec_t);

  // Vectors for holding the host-side (CPU-side) data
  std::vector<vec_t> bucket_a_data;
  bucket_a_data.reserve(bucket_a_bytes);
  std::vector<vec_hash_t> bucket_c_data;
  bucket_c_data.reserve(bucket_c_bytes);

  // Initialize each vector
  for (int i = 0; i < num_elements; i++) {
    bucket_a_data.push_back(bucket_a[i]);
    bucket_c_data.push_back(bucket_c[i]);
  }

  // Allocate memory on the device
  vec_t *d_bucket_a;
  vec_hash_t *d_bucket_c;
  vec_t *d_result;
  hipMalloc(&d_bucket_a, bucket_a_bytes);
  hipMalloc(&d_bucket_c, bucket_c_bytes);
  hipMalloc(&d_result, result_bytes);

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(d_bucket_a, bucket_a_data.data(), bucket_a_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_bucket_c, bucket_c_data.data(), bucket_c_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_result, result.data(), result_bytes, hipMemcpyHostToDevice);

  // Threads per CTA
  int THREADS = 128;

  // Blocks per grid dimension
  int BLOCKS = num_elements / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  sketch_query<<<blocks, threads>>>(d_bucket_a, d_bucket_c, d_result, num_guesses, seed);

  // Copy back to the host
  hipMemcpy(result.data(), d_result, result_bytes, hipMemcpyDeviceToHost);

  // Free memory on device
  hipFree(d_bucket_a);
  hipFree(d_bucket_c);
  hipFree(d_result);
}