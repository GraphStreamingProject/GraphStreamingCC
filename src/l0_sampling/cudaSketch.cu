#include "hip/hip_runtime.h"
#include "../../include/l0_sampling/cudaSketch.cuh"

#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include <iterator>

/*__host__ __device__  col_hash_t bucket_col_index_hash(const vec_t& update_idx, const long seed_and_col) {
  return col_hash(&update_idx, sizeof(update_idx), seed_and_col);
}

__host__ __device__  vec_hash_t bucket_index_hash(const vec_t& index, long sketch_seed) {
  return vec_hash(&index, sizeof(index), sketch_seed);
}*/

__host__ __device__  bool bucket_contains(const col_hash_t& col_index_hash, const col_hash_t& guess_nonzero) {
  return (col_index_hash & guess_nonzero) == 0; // use guess_nonzero (power of 2) to check ith bit
}

/*__host__ __device__ bool bucket_is_good(const vec_t& a, const vec_hash_t& c, const unsigned bucket_col, const vec_t& guess_nonzero, const long& sketch_seed) {
  return c == bucket_index_hash(a, sketch_seed)
    && bucket_contains(bucket_col_index_hash(a, sketch_seed + bucket_col), guess_nonzero);
}*/

__global__ void sketch_update(vec_t* combined_memory, const size_t num_elems, const size_t num_buckets, const size_t num_guesses, const uint64_t seed, 
                              const vec_t update_idx, const vec_hash_t update_hash) {

    int currentId = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(currentId < num_buckets) {
      for (int guessId = 0; guessId < num_guesses; guessId++) {
        unsigned bucket_id = currentId * num_guesses + guessId;
        if (bucket_contains(combined_memory[(2 * num_elems) + currentId], ((col_hash_t)1) << guessId)){
          combined_memory[bucket_id] = combined_memory[bucket_id] ^ update_idx;
          combined_memory[bucket_id + num_elems] = combined_memory[bucket_id + num_elems] ^ update_hash;
        }
        else {
          return;
        }
      }
    }
}

/*__global__ void sketch_query(const vec_t* bucket_a, const vec_hash_t* bucket_c, vec_t* result, size_t num_guesses, uint64_t seed) {
    int bucketId = blockIdx.x * blockDim.x + threadIdx.x;

    for (int guessId = 0; guessId < num_guesses; guessId++) {
      unsigned bucket_id = bucketId * num_guesses + guessId;
      // Check if bucket is good
    }
}*/

void bucket_update(vec_t& a, vec_t& c, const vec_t& update_idx, const vec_hash_t& update_hash) {
  a ^= update_idx;
  c ^= update_hash;
}

CudaSketch::CudaSketch(size_t numElems, size_t numBuckets, size_t numGuesses, uint64_t currentSeed) {
  num_elems = numElems;
  num_buckets = numBuckets;
  num_guesses = numGuesses;
  seed = currentSeed;
};

void CudaSketch::update(vec_t* &combined_memory, vec_t* &combined_device_memory, const vec_t& update_idx) {

  vec_hash_t update_hash = Bucket_Boruvka::index_hash(update_idx, seed);
  bucket_update(combined_memory[num_elems - 1], combined_memory[(num_elems * 2) - 1], update_idx, update_hash);

  for (unsigned i = 0; i < num_buckets; ++i) {
    combined_memory[i + (num_elems * 2)] = Bucket_Boruvka::col_index_hash(update_idx, seed + i);
  }

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(combined_device_memory, combined_memory, (2 * (num_elems * sizeof(vec_t))) + (num_buckets * sizeof(col_hash_t)), hipMemcpyHostToDevice);

  // Threads per CTA
  int num_threads = 1 << 10;

  // Blocks per grid dimension[i] = 0;
  int num_blocks = (num_buckets + num_threads - 1) / num_threads;

  //dim3 threads(num_threads, num_threads);
  //dim3 blocks(num_blocks, num_blocks);

  // Launch kernel
  //sketch_update<<<num_blocks, num_threads>>>(d_bucket_a, d_bucket_c, num_buckets, num_guesses, seed, d_col_index_hashes, update_idx, update_hash);
  sketch_update<<<num_blocks, num_threads>>>(combined_device_memory, num_elems, num_buckets, num_guesses, seed, update_idx, update_hash);

  hipMemcpy(combined_memory, combined_device_memory, (2 * (num_elems * sizeof(vec_t))) + (num_buckets * sizeof(col_hash_t)), hipMemcpyDeviceToHost);
}

/*void CudaSketch::query() {
  vec_t bucket_a_bytes = sizeof(vec_t) * num_elements;
  vec_hash_t bucket_c_bytes = sizeof(vec_hash_t) * num_elements;
  vec_t result_bytes = sizeof(vec_t);

  // Vectors for holding the host-side (CPU-side) data
  std::vector<vec_t> bucket_a_data;
  bucket_a_data.reserve(bucket_a_bytes);
  std::vector<vec_hash_t> bucket_c_data;
  bucket_c_data.reserve(bucket_c_bytes);

  // Initialize each vector
  for (int i = 0; i < num_elements; i++) {
    bucket_a_data.push_back(bucket_a[i]);
    bucket_c_data.push_back(bucket_c[i]);
  }

  // Allocate memory on the device
  vec_t *d_bucket_a;
  vec_hash_t *d_bucket_c;
  vec_t *d_result;
  hipMalloc(&d_bucket_a, bucket_a_bytes);
  hipMalloc(&d_bucket_c, bucket_c_bytes);
  hipMalloc(&d_result, result_bytes);

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(d_bucket_a, bucket_a_data.data(), bucket_a_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_bucket_c, bucket_c_data.data(), bucket_c_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_result, result.data(), result_bytes, hipMemcpyHostToDevice);

  // Threads per CTA
  int num_threads = 128;

  // Blocks per grid dimension
  int num_blocks = ceil(num_buckets / num_threads);

  // Launch kernel
  sketch_query<<<num_blocks, num_threads>>>(d_bucket_a, d_bucket_c, d_result, num_guesses, seed);

  // Copy back to the host
  hipMemcpy(result.data(), d_result, result_bytes, hipMemcpyDeviceToHost);

  // Free memory on device
  hipFree(d_bucket_a);
  hipFree(d_bucket_c);
  hipFree(d_result);
}*/