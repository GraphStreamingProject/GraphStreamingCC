#include "hip/hip_runtime.h"
#include <vector>
#include <graph.h>
#include <graph_worker.h>
#include <cuda_graph.h>
#include <map>
#include <binary_graph_stream.h>
#include "../src/cuda_kernel.cu"

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: stream_file, graph_workers, reader_threads" << std::endl;
    exit(EXIT_FAILURE);
  }

  std::string stream_file = argv[1];
  int num_threads = std::atoi(argv[2]);
  if (num_threads < 1) {
    std::cout << "ERROR: Invalid number of graph workers! Must be > 0." << std::endl;
    exit(EXIT_FAILURE);
  }
  int reader_threads = std::atoi(argv[3]);

  BinaryGraphStream_MT stream(stream_file, 1024*32);
  node_id_t num_nodes = stream.nodes();
  size_t num_updates  = stream.edges();
  std::cout << "Running process_stream with CUDA: " << std::endl;
  std::cout << "Processing stream: " << stream_file << std::endl;
  std::cout << "nodes       = " << num_nodes << std::endl;
  std::cout << "num_updates = " << num_updates << std::endl;
  std::cout << std::endl;

  CudaGraph cudaGraph;

  auto config = GraphConfiguration().gutter_sys(STANDALONE).num_groups(num_threads);
  config.gutter_conf().gutter_factor(-4);
  Graph g{num_nodes, config, &cudaGraph, reader_threads};

  Supernode** supernodes;
  supernodes = g.getSupernodes();

  // Get variable from sample supernode
  int num_sketches = supernodes[0]->get_num_sktch();
  
  // Get variables from sample sketch
  size_t num_elems = supernodes[0]->get_sketch(0)->get_num_elems();
  size_t num_columns = supernodes[0]->get_sketch(0)->get_columns();
  size_t num_guesses = supernodes[0]->get_sketch(0)->get_num_guesses();
  
  CudaUpdateParams* cudaUpdateParams;
  gpuErrchk(hipMallocManaged(&cudaUpdateParams, sizeof(CudaUpdateParams)));
  cudaUpdateParams[0] = CudaUpdateParams(num_nodes, num_updates, num_sketches, num_elems, num_columns, num_guesses);

  std::cout << "num_sketches: " << num_sketches << "\n";
  std::cout << "num_elems: " << num_elems << "\n";
  std::cout << "num_columns: " << num_columns << "\n";
  std::cout << "num_guesses: " << num_guesses << "\n";

  CudaCCParams* cudaCCParams;
  gpuErrchk(hipMallocManaged(&cudaCCParams, sizeof(CudaCCParams)));
  cudaCCParams[0] = CudaCCParams(num_nodes, num_sketches, num_elems, num_columns, num_guesses);

  std::pair<Edge, SampleSketchRet> *graph_query = new std::pair<Edge, SampleSketchRet>[num_nodes];

  // Hashmap that stores node ids and edge ids that need to be updated
  std::map<int, std::vector<vec_t>> graphUpdates;
  std::vector<node_id_t> graph_reps(num_nodes);

  // Start timer for initializing
  auto init_start = std::chrono::steady_clock::now();

  for (int i = 0; i < num_nodes; i++) {
    // Initialize each key in graphUpdates with empty vector
    graphUpdates[i] = std::vector<vec_t>{};
    
    // Initialize cudaUpdateParams
    cudaUpdateParams[0].nodeNumUpdates[i] = 0;
    cudaUpdateParams[0].nodeStartIndex[i] = 0;

    // Initialize cudaCCParams
    cudaCCParams[0].reps[i] = i;
    cudaCCParams[0].temp_reps[i] = 0;
    cudaCCParams[0].query[i] = {1, ZERO};
    cudaCCParams[0].parent[i] = i;
    cudaCCParams[0].size[i] = 1;

    cudaCCParams[0].sample_idxs[i] = supernodes[i]->curr_idx();
    cudaCCParams[0].merged_sketches[i] = supernodes[i]->get_merged_sketches();

    graph_reps[i] = i;
  }

  for (int i = 0; i < num_updates * 2; i++) {
    cudaUpdateParams[0].edgeUpdates[i] = 0;
  }

  std::cout << "Finished initializing CUDA parameters\n";
  auto init_end = std::chrono::steady_clock::now();
  std::chrono::duration<double> init_time = init_end - init_start;
  std::cout << "CUDA parameters init duration: " << init_time.count() << std::endl;

  cudaGraph.configure(cudaUpdateParams[0].edgeUpdates, cudaUpdateParams[0].nodeNumUpdates, cudaUpdateParams[0].nodeStartIndex, num_nodes);
  
  GutteringSystem *gts = g.getGTS();
  MT_StreamReader reader(stream);
  GraphUpdate upd;

  // Start timer for reordering
  auto reorder_start = std::chrono::steady_clock::now();

  // Insert an edge to guttering system
  for (size_t e = 0; e < num_updates; e++) {
    upd = reader.get_edge();
    Edge &edge = upd.edge;

    gts->insert({edge.src, edge.dst});
    std::swap(edge.src, edge.dst);
    gts->insert({edge.src, edge.dst});
  }

  gts->force_flush();
  GraphWorker::pause_workers();
  cudaGraph.fillParam();

  std::cout << "Finished Reordering using GTS\n";
  auto reorder_end = std::chrono::steady_clock::now();
  std::chrono::duration<double> reorder_time = reorder_end - reorder_start;
  std::cout << "Reordering duration: " << reorder_time.count() << std::endl;

  CudaSketch* cudaSketches;
  gpuErrchk(hipMallocManaged(&cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch)));

  long* sketchSeeds;
  gpuErrchk(hipMallocManaged(&sketchSeeds, num_nodes * num_sketches * sizeof(long)));

  // Allocate space for all buckets
  vec_t* d_bucket_a;
  vec_hash_t* d_bucket_c;
  gpuErrchk(hipMallocManaged(&d_bucket_a, (num_nodes * num_sketches * num_elems * sizeof(vec_t))));
  gpuErrchk(hipMallocManaged(&d_bucket_c, (num_nodes * num_sketches * num_elems * sizeof(vec_hash_t))));

  for (int i = 0; i < (num_nodes * num_sketches * num_elems); i++) {
    d_bucket_a[i] = 0;
    d_bucket_c[i] = 0;
  }

  // Create a vector of cuda supernodes and sketches
  for (int i = 0; i < num_nodes; i++) {
    for (int j = 0; j < num_sketches; j++) {
      Sketch* sketch = supernodes[i]->get_sketch(j);

      int bucket_id = (i * num_sketches * num_elems) + (j * num_elems);
      vec_t* bucket_a = &d_bucket_a[bucket_id];
      vec_hash_t* bucket_c = &d_bucket_c[bucket_id];

      // Rewrite sketch's bucket_a and bucket_c memory location
      sketch->set_bucket_a(bucket_a);
      sketch->set_bucket_c(bucket_c);

      CudaSketch cudaSketch(bucket_a, bucket_c, sketch->get_seed());
      cudaSketches[(i * num_sketches) + j] = cudaSketch;
      sketchSeeds[(i * num_sketches) + j] = sketch->get_seed();
    }
  }

  // Number of threads
  int num_device_threads = 1024;
  
  // Number of blocks
  int num_device_blocks = num_nodes;

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";

  std::cout << "Allocated Shared Memory of: " << (num_elems * num_sketches * sizeof(vec_t_cu)) + (num_elems * num_sketches * sizeof(vec_hash_t)) << "\n";

  // Prefetch memory to device 
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].edgeUpdates, num_updates * sizeof(vec_t) * 2, device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].nodeNumUpdates, num_nodes * sizeof(node_id_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].nodeStartIndex, num_nodes * sizeof(node_id_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch), device_id));
  gpuErrchk(hipMemPrefetchAsync(sketchSeeds, num_nodes * num_sketches * sizeof(long), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_a, num_nodes * num_sketches * num_elems * sizeof(vec_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_c, num_nodes * num_sketches * num_elems * sizeof(vec_hash_t), device_id));

  // Start timer for kernel
  auto ins_start = std::chrono::steady_clock::now();

  // Call kernel code
  std::cout << "Update Kernel Starting...\n";
  streamUpdate(num_device_threads, num_device_blocks, cudaUpdateParams, cudaSketches, sketchSeeds);
  std::cout << "Update Kernel finished.\n";

  // End timer for kernel
  auto ins_end = std::chrono::steady_clock::now();
  
  // Update graph's num_updates value
  g.num_updates += num_updates * 2;

  // Start timer for cc
  auto cc_start = std::chrono::steady_clock::now();

  bool first_round = true;
  int round_num = 0;

  std::vector<std::chrono::duration<double>> round_durations;
  std::vector<std::chrono::duration<double>> sample_durations;
  std::vector<std::chrono::duration<double>> to_merge_durations;
  std::vector<std::chrono::duration<double>> merge_durations;

  // Prepare graph's size and parent pointers
  g.fillSize(1);

  for (node_id_t i = 0; i < num_nodes; ++i) {
    g.setParent(i, i);
  }

  // Start sampling supernodes
  do {
    // Start timer for initial time for round
    auto round_start = std::chrono::steady_clock::now();

    cudaCCParams[0].modified[0] = false;
    g.setModified(false);

    // Number of blocks
    num_device_blocks = (cudaCCParams[0].num_nodes[0] + num_device_threads - 1) / num_device_threads;

    // Get and check sample_idx of each supernodes
    for (int i = 0; i < cudaCCParams[0].num_nodes[0]; i++) {
      int index = cudaCCParams[0].reps[i];

      if(cudaCCParams[0].sample_idxs[index] >= cudaCCParams[0].merged_sketches[index]) throw OutOfQueriesException();

      Sketch* sketch = supernodes[index]->get_sketch(cudaCCParams[0].sample_idxs[index]);

      // Check if this sketch has already been queried
      if(sketch->get_queried()) throw MultipleQueryException();
      
      sketch->set_queried(true);

      // Increment current supernode's sample idx
      cudaCCParams[0].sample_idxs[index]++;
    }

    // Start timer for sampling
    auto sample_start = std::chrono::steady_clock::now();

    // Sample each supernodes
    cuda_sample_supernodes(num_device_threads, num_device_blocks, cudaCCParams, cudaSketches);

    // End timer for sampling
    auto sample_end = std::chrono::steady_clock::now();
    sample_durations.push_back(sample_end - sample_start);

    // Start timer for to_merge
    auto to_merge_start = std::chrono::steady_clock::now();

    // Reset to_merge
    /*for(int i = 0; i < num_nodes; i++) {
      cudaCCParams[0].temp_reps[i] = 0;

      for (int j = 0; j < cudaCCParams[0].to_merge[i].size[0]; j++) {
        cudaCCParams[0].to_merge[i].children[j] = 0;
      }
      cudaCCParams[0].to_merge[i].size[0] = 0;
    }

    cuda_supernodes_to_merge(num_device_threads, num_device_blocks, cudaCCParams);

    std::cout << "Reps: ";
    for(int i = 0; i < cudaCCParams[0].num_nodes[0]; i++) {
      std::cout << cudaCCParams[0].reps[i] << " ";
    }
    std::cout << "\n";*/

    for (int i = 0; i < cudaCCParams[0].num_nodes[0]; i++) {
      int index = graph_reps[i];
      graph_query[index] = {cudaCCParams[0].query[index].edge, cudaCCParams[0].query[index].ret_code};
    }

    std::vector<std::vector<node_id_t>> to_merge = g.supernodes_to_merge(graph_query, graph_reps);

    cudaCCParams[0].num_nodes[0] = graph_reps.size();
    for (int i = 0; i < graph_reps.size(); i++) {
      cudaCCParams[0].reps[i] = graph_reps[i];
    }
    cudaCCParams[0].modified[0] = g.getModified();

    // End timer for to_merge
    auto to_merge_end = std::chrono::steady_clock::now();
    to_merge_durations.push_back(to_merge_end - to_merge_start);

    // Start timer for merge
    auto merge_start = std::chrono::steady_clock::now();

    // Transfer to_merge information
    for (int i = 0; i < num_nodes; i++) {
      cudaCCParams[0].to_merge[i].size[0] = to_merge[i].size();
      for (int j = 0; j < to_merge[i].size(); j++) {
        cudaCCParams[0].to_merge[i].children[j] = to_merge[i][j];
      }
    } 

    cuda_merge_supernodes(num_device_threads, num_device_blocks, cudaCCParams, cudaSketches);

    // End timer for merge
    auto merge_end = std::chrono::steady_clock::now();
    merge_durations.push_back(merge_end - merge_start);

    first_round = false;
    round_num++;

    // End timer for round
    auto round_end = std::chrono::steady_clock::now();
    round_durations.push_back(round_end - round_start);

  } while (cudaCCParams[0].modified[0]);

  /*for (node_id_t i = 0; i < num_nodes; ++i) {
    g.setSize(i, cudaCCParams[0].size[i]);
    g.setParent(i, cudaCCParams[0].parent[i]);
  }*/

  // Find connected components
  auto CC_num = g.cc_from_dsu().size();

  // End timer for cc
  auto cc_end = std::chrono::steady_clock::now();

  std::chrono::duration<double> insert_time = ins_end - ins_start;
  std::chrono::duration<double> cc_time = cc_end - cc_start;

  double num_seconds = insert_time.count();
  std::cout << "Total insertion time(sec):    " << num_seconds << std::endl;
  std::cout << "Updates per second:           " << stream.edges() / num_seconds << std::endl;
  std::cout << "Total CC query latency:       " << cc_time.count() << std::endl;

  for (int i = 0; i < sample_durations.size(); i++) {
    std::cout << "    Round " << i << ":                  " << round_durations[i].count() << std::endl;
    std::cout << "        Sampling:               " << sample_durations[i].count() << std::endl;
    std::cout << "        To_Merge:               " << to_merge_durations[i].count() << std::endl;
    std::cout << "        Merge:                  " << merge_durations[i].count() << std::endl;
  }
  std::cout << "Connected Components:         " << CC_num << std::endl;
}
