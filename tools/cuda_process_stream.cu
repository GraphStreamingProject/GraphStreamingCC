#include <vector>
#include <graph.h>
#include <map>
#include <binary_graph_stream.h>
#include "../src/cuda_kernel.cu"

int main(int argc, char **argv) {
  if (argc != 3) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: stream_file, graph_workers" << std::endl;
  }

  std::string stream_file = argv[1];
  int num_threads = std::atoi(argv[2]);
  if (num_threads < 1) {
    std::cout << "ERROR: Invalid number of graph workers! Must be > 0." << std::endl;
  }

  BinaryGraphStream_MT stream(stream_file, 1024*32);
  node_id_t num_nodes = stream.nodes();
  size_t num_updates  = stream.edges();
  std::cout << "Running process_stream with CUDA: " << std::endl;
  std::cout << "Processing stream: " << stream_file << std::endl;
  std::cout << "nodes       = " << num_nodes << std::endl;
  std::cout << "num_updates = " << num_updates << std::endl;
  std::cout << std::endl;

  auto config = GraphConfiguration().gutter_sys(STANDALONE).num_groups(num_threads);
  config.gutter_conf().gutter_factor(-4);
  Graph g{num_nodes, config, 1};

  Supernode** supernodes;
  supernodes = g.getSupernodes();

  // Get variable from sample supernode
  int num_sketches = supernodes[0]->get_num_sktch();
  
  // Get variables from sample sketch
  size_t num_elems = supernodes[0]->get_sketch(0)->get_num_elems();
  size_t num_columns = supernodes[0]->get_sketch(0)->get_columns();
  size_t num_guesses = supernodes[0]->get_sketch(0)->get_num_guesses();
  
  CudaUpdateParams* cudaUpdateParams;
  gpuErrchk(hipMallocManaged(&cudaUpdateParams, sizeof(CudaUpdateParams)));
  cudaUpdateParams[0] = CudaUpdateParams(num_nodes, num_updates, num_sketches, num_elems, num_columns, num_guesses);

  CudaCCParams* cudaCCParams;
  gpuErrchk(hipMallocManaged(&cudaCCParams, sizeof(CudaCCParams)));
  cudaCCParams[0] = CudaCCParams(num_nodes, num_sketches, num_elems, num_columns, num_guesses);

  std::pair<Edge, SampleSketchRet> *graph_query = new std::pair<Edge, SampleSketchRet>[num_nodes];
  std::vector<node_id_t> graph_reps(num_nodes);

  // Hashmap that stores node ids and edge ids that need to be updated
  std::map<int, std::vector<vec_t>> graphUpdates;

  for (int i = 0; i < num_nodes; i++) {
    // Initialize each key in graphUpdates with empty vector
    graphUpdates[i] = std::vector<vec_t>{};
    
    // Initialize cudaUpdateParams
    cudaUpdateParams[0].nodeNumUpdates[i] = 0;
    cudaUpdateParams[0].nodeStartIndex[i] = 0;

    // Initialize cudaCCParams
    cudaCCParams[0].reps[i] = i;
    cudaCCParams[0].query[i] = {1, ZERO};
    cudaCCParams[0].sketchIds[i] = 0;

    // Initialize query and reps for sending to graph class
    graph_reps[i] = i;
  }
  
  MT_StreamReader reader(stream);
  GraphUpdate upd;

  // Collect all the edges that need to be updated
  for (size_t e = 0; e < num_updates; e++) {
    upd = reader.get_edge();
    Edge &edge = upd.edge;

    graphUpdates[edge.src].push_back(static_cast<vec_t>(concat_pairing_fn(edge.src, edge.dst)));
    graphUpdates[edge.dst].push_back(static_cast<vec_t>(concat_pairing_fn(edge.dst, edge.src)));   
  }

  std::cout << "Finished initializing graphUpdates\n";

  // Transfer graphUpdates to nodeUpdates and edgeUpdates
  int nodeIt = 0;
  int startIndex = 0;
  for (auto it = graphUpdates.begin(); it != graphUpdates.end(); it++) {
    cudaUpdateParams[0].nodeStartIndex[it->first] = startIndex;
    cudaUpdateParams[0].nodeNumUpdates[it->first] = it->second.size();
    for (int i = 0; i < it->second.size(); i++) {
      cudaUpdateParams[0].nodeUpdates[nodeIt] = it->first;
      cudaUpdateParams[0].edgeUpdates[nodeIt] = it->second.at(i);
      nodeIt++;
    }
    startIndex += it->second.size();
  }

  std::cout << "Finished initializing nodeUpdates and edgeUpdates\n";

  CudaSketch* cudaSketches;
  gpuErrchk(hipMallocManaged(&cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch)));

  long* sketchSeeds;
  gpuErrchk(hipMallocManaged(&sketchSeeds, num_nodes * num_sketches * sizeof(long)));

  // Allocate space for all buckets
  vec_t* d_bucket_a;
  vec_hash_t* d_bucket_c;
  gpuErrchk(hipMallocManaged(&d_bucket_a, (num_nodes * num_sketches * num_elems * sizeof(vec_t))));
  gpuErrchk(hipMallocManaged(&d_bucket_c, (num_nodes * num_sketches * num_elems * sizeof(vec_hash_t))));

  for (int i = 0; i < (num_nodes * num_sketches * num_elems); i++) {
    d_bucket_a[i] = 0;
    d_bucket_c[i] = 0;
  }

  // Create a vector of cuda supernodes and sketches
  for (int i = 0; i < num_nodes; i++) {
    for (int j = 0; j < num_sketches; j++) {
      Sketch* sketch = supernodes[i]->get_sketch(j);

      int bucket_id = (i * num_sketches * num_elems) + (j * num_elems);
      vec_t* bucket_a = &d_bucket_a[bucket_id];
      vec_hash_t* bucket_c = &d_bucket_c[bucket_id];

      // Rewrite sketch's bucket_a and bucket_c memory location
      sketch->set_bucket_a(bucket_a);
      sketch->set_bucket_c(bucket_c);

      CudaSketch cudaSketch(bucket_a, bucket_c, sketch->get_seed());
      cudaSketches[(i * num_sketches) + j] = cudaSketch;
      sketchSeeds[(i * num_sketches) + j] = sketch->get_seed();
    }
  }

  // Number of threads
  int num_device_threads = 1024;
  
  // Number of blocks
  int num_device_blocks = num_nodes;

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";

  std::cout << "Allocated Shared Memory of: " << (num_elems * num_sketches * sizeof(vec_t_cu)) + (num_elems * num_sketches * sizeof(vec_hash_t)) << "\n";

  // Prefetch memory to device 
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].nodeUpdates, num_updates * sizeof(node_id_t) * 2, device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].edgeUpdates, num_updates * sizeof(vec_t) * 2, device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].nodeNumUpdates, num_nodes * sizeof(node_id_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaUpdateParams[0].nodeStartIndex, num_nodes * sizeof(node_id_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch), device_id));
  gpuErrchk(hipMemPrefetchAsync(sketchSeeds, num_nodes * num_sketches * sizeof(long), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_a, num_nodes * num_sketches * num_elems * sizeof(vec_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_c, num_nodes * num_sketches * num_elems * sizeof(vec_hash_t), device_id));

  // Start timer for kernel
  auto ins_start = std::chrono::steady_clock::now();

  // Call kernel code
  std::cout << "Update Kernel Starting...\n";
  streamUpdate(num_device_threads, num_device_blocks, cudaUpdateParams, cudaSketches, sketchSeeds);
  std::cout << "Update Kernel finished.\n";

  // End timer for kernel
  auto ins_end = std::chrono::steady_clock::now();
  
  // Update graph's num_updates value
  g.num_updates += num_updates * 2;

  // Start timer for cc
  auto cc_start = std::chrono::steady_clock::now();

  bool first_round = true;
  Supernode** copy_supernodes;
  std::vector<std::chrono::duration<double>> sample_durations;

  // Prepare graph's size and parent pointers
  g.fillSize(1);

  for (node_id_t i = 0; i < num_nodes; ++i) {
    g.setParent(i, i);
  }

  // Start sampling supernodes
  do {
    g.setModified(false);

    cudaCCParams[0].num_nodes = graph_reps.size();

    // Number of blocks
    num_device_blocks = (cudaCCParams[0].num_nodes + num_device_threads - 1) / num_device_threads;

    // Get and check sample_idx of each supernodes
    for (int i = 0; i < cudaCCParams[0].num_nodes; i++) {
      int index = graph_reps[i];
      cudaCCParams[0].reps[i] = index;

      if (supernodes[index]->out_of_queries()) throw OutOfQueriesException();

      int curr_idx = supernodes[index]->curr_idx();

      cudaCCParams[0].sketchIds[i] = curr_idx;
      Sketch* sketch = supernodes[index]->get_sketch(curr_idx);

      // Check if this sketch has already been queried
      if(sketch->get_queried()) throw MultipleQueryException();
      
      sketch->set_queried(true);

      // Increment current supernode's sample idx
      supernodes[index]->inc_sample_idx();
    }

    // Start timer for sampling
    auto sample_start = std::chrono::steady_clock::now();

    // Sample each supernodes
    cuda_sample_supernodes(num_device_threads, num_device_blocks, cudaCCParams, cudaSketches);

    // End timer for sampling
    auto sample_end = std::chrono::steady_clock::now();
    sample_durations.push_back(sample_end - sample_start);

    int count = 0;
    for (int i = 0; i < cudaCCParams[0].num_nodes; i++) {
      int index = graph_reps[i];
      graph_query[index] = {inv_concat_pairing_fn(cudaCCParams[0].query[index].non_zero), cudaCCParams[0].query[index].ret_code};
    }

    std::vector<std::vector<node_id_t>> to_merge = g.supernodes_to_merge(graph_query, graph_reps);

    g.merge_supernodes(copy_supernodes, graph_reps, to_merge, first_round && false);

    first_round = false;

  } while (g.getModified());

  // Find connected components
  auto CC_num = g.cc_from_dsu().size();

  // End timer for cc
  auto cc_end = std::chrono::steady_clock::now();

  std::chrono::duration<double> insert_time = ins_end - ins_start;
  std::chrono::duration<double> cc_time = cc_end - cc_start;

  double num_seconds = insert_time.count();
  std::cout << "Total insertion time(sec):    " << num_seconds << std::endl;
  std::cout << "Updates per second:           " << stream.edges() / num_seconds << std::endl;
  std::cout << "Total CC query latency:       " << cc_time.count() << std::endl;

  for (int i = 0; i < sample_durations.size(); i++) {
    std::cout << "    Sample " << i << ":                 " << sample_durations[i].count() << std::endl;
  }
  std::cout << "Connected Components:         " << CC_num << std::endl;
}
