#include "hip/hip_runtime.h"
#include <vector>
#include <map>
#include <random>
#include <fstream>
#include <string>
#include <cmath>

#include <graph.h>
#include <graph_worker.h>
#include <binary_graph_stream.h>
#include <cuda_graph.cuh>
#include <mincut_graph.h>

constexpr double epsilon = 0.2;

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: stream_file, graph_workers, reader_threads" << std::endl;
    exit(EXIT_FAILURE);
  }

  std::string stream_file = argv[1];
  int num_threads = std::atoi(argv[2]);
  if (num_threads < 1) {
    std::cout << "ERROR: Invalid number of graph workers! Must be > 0." << std::endl;
    exit(EXIT_FAILURE);
  }
  int reader_threads = std::atoi(argv[3]);

  BinaryGraphStream_MT stream(stream_file, 1024*32);
  node_id_t num_nodes = stream.nodes();
  size_t num_updates  = stream.edges();
  std::cout << "Running process_stream with CUDA: " << std::endl;
  std::cout << "Processing stream: " << stream_file << std::endl;
  std::cout << "nodes       = " << num_nodes << std::endl;
  std::cout << "num_updates = " << num_updates << std::endl;
  std::cout << std::endl;

  int k = log2(num_nodes) / (epsilon * epsilon);

  std::cout << "epsilon: " << epsilon << std::endl;
  std::cout << "k: " << k << std::endl;

  int num_graphs = 1 + (int)(2 * log2(num_nodes));
  std::cout << "Total num_graphs: " << num_graphs << "\n";

  auto config = GraphConfiguration().gutter_sys(CACHETREE).num_groups(num_threads);
  // Configuration is from cache_exp.cpp
  config.gutter_conf().page_factor(1)
              .buffer_exp(20)
              .fanout(64)
              .queue_factor(8)
              .num_flushers(2)
              .gutter_bytes(32 * 1024)
              .wq_batch_per_elm(8);

  CudaGraph cudaGraph;
  MinCutGraph* graphs[num_graphs];

  for (int i = 0; i < num_graphs; i++) {
    if (i == 0) {
      graphs[i] = new MinCutGraph{num_nodes, config, &cudaGraph, k, reader_threads};
    }
    else {
      // Reuse the GTS made from graphs[0]
      graphs[i] = new MinCutGraph{num_nodes, config, graphs[0]->getGTS(), &cudaGraph, k, reader_threads};
    }
  }

  double total_graphs_sketch_size = graphs[0]->getTotalSketchSize() * num_graphs;

  if(total_graphs_sketch_size > 1000000000) {
    std::cout << "Total Graphs Sketch Memory Size: " << total_graphs_sketch_size / 1000000000 << " GB\n";
  }
  else if(total_graphs_sketch_size > 1000000) {
    std::cout << "Total Graphs Sketch Memory Size: " << total_graphs_sketch_size / 1000000 << " MB\n";
  }
  else {
    std::cout << "Total Graphs Sketch Memory Size: " << total_graphs_sketch_size / 1000 << " KB\n";
  }

  Supernode** supernodes;
  supernodes = graphs[0]->getSupernodes();

  // Get variable from sample supernode
  int num_sketches = supernodes[0]->get_num_sktch();

  // Get variables from sample sketch
  size_t num_elems = supernodes[0]->get_sketch(0)->get_num_elems();
  size_t num_columns = supernodes[0]->get_sketch(0)->get_columns();
  size_t num_guesses = supernodes[0]->get_sketch(0)->get_num_guesses();

  std::cout << "num_sketches: " << num_sketches << "\n";
  std::cout << "num_elems: " << num_elems << "\n";
  std::cout << "num_columns: " << num_columns << "\n";
  std::cout << "num_guesses: " << num_guesses << "\n";

  // Start timer for initializing
  auto init_start = std::chrono::steady_clock::now();

  GutteringSystem *gts = graphs[0]->getGTS();
  int batch_size = gts->gutter_size() / sizeof(node_id_t);
  int stream_multiplier = 4;

  std::cout << "Batch_size: " << batch_size << "\n";
  
  CudaUpdateParams** cudaUpdateParams;
  gpuErrchk(hipMallocManaged(&cudaUpdateParams, sizeof(CudaUpdateParams*) * num_graphs));
  for (int i = 0; i < num_graphs; i++) {
    cudaUpdateParams[i] = new CudaUpdateParams(num_nodes, num_updates, num_sketches, num_elems, num_columns, num_guesses, num_threads, batch_size, stream_multiplier, k);
  }

  std::cout << "Initialized cudaUpdateParams\n";

  long* sketchSeeds;
  size_t sketch_width = Sketch::column_gen(Sketch::get_failure_factor());
  gpuErrchk(hipMallocManaged(&sketchSeeds, num_graphs * num_nodes * num_sketches * k * sizeof(long)));

  // Initialize sketch seeds
  std::vector<Supernode**> all_supernodes;
  
  for (int graph_id = 0; graph_id < num_graphs; graph_id++) {
    Supernode** graph_supernodes;
    graph_supernodes = graphs[graph_id]->getSupernodes();
    for (int node_id = 0; node_id < num_nodes; node_id++) {
      for (int k_id = 0; k_id < k; k_id++) {
        for (int j = 0; j < num_sketches; j++) {
          Sketch* sketch = graph_supernodes[(node_id * k) + k_id]->get_sketch(j);
          sketchSeeds[(graph_id * num_nodes * num_sketches * k) + (node_id * num_sketches * k) + (k_id * num_sketches) + j] = sketch->get_seed();
        }
      }
    }
    all_supernodes.push_back(graph_supernodes);
  }

  int maxBytes = num_elems * num_sketches * sizeof(vec_t_cu) + num_elems * num_sketches * sizeof(vec_hash_t);
  cudaGraph.cudaKernel.kernelUpdateSharedMemory(maxBytes);

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  struct hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device_id);
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";
  std::cout << "Maximum Shared Memory per block: " << props.sharedMemPerBlock << " bytes\n";
  std::cout << "Maximum Shared Memory per block Optin: " << props.sharedMemPerBlockOptin << " bytes\n";

  cudaGraph.k_configure(cudaUpdateParams, all_supernodes.data(), sketchSeeds, num_threads, k, num_graphs);

  std::cout << "Allocated Shared Memory of: " << maxBytes << "\n";

  // Prefetch memory to device  
  gpuErrchk(hipMemPrefetchAsync(sketchSeeds, num_graphs * num_nodes * num_sketches * k * sizeof(long), device_id));

  MT_StreamReader reader(stream);
  GraphUpdate upd;
  
  std::cout << "Finished initializing CUDA parameters\n";
  std::chrono::duration<double> init_time = std::chrono::steady_clock::now() - init_start;
  std::cout << "CUDA parameters init duration: " << init_time.count() << std::endl;

  // Start timer for kernel
  auto ins_start = std::chrono::steady_clock::now();

  // Call kernel code
  std::cout << "Update Kernel Starting...\n";

  // Do the edge updates
  std::vector<std::thread> threads;
  threads.reserve(reader_threads);
  auto task = [&](const int thr_id) {
    MT_StreamReader reader(stream);
    GraphUpdate upd;
    while(true) {
      upd = reader.get_edge();
      if (upd.type == BREAKPOINT) break;
      Edge &edge = upd.edge;

      gts->insert({edge.src, edge.dst}, thr_id);
      std::swap(edge.src, edge.dst);
      gts->insert({edge.src, edge.dst}, thr_id);
    }
  };

  // start inserters
  for (int t = 0; t < reader_threads; t++) {
    threads.emplace_back(task, t);
  }
  // wait for inserters to be done
  for (int t = 0; t < reader_threads; t++) {
    threads[t].join();
  }

  std::cout << "  Flush Starting...\n";
  
  auto flush_start = std::chrono::steady_clock::now();
  gts->force_flush();
  GraphWorker::pause_workers();
  hipDeviceSynchronize();

  std::cout << "  Applying Flush Updates...\n";

  cudaGraph.k_applyFlushUpdates();

  for(int i = 0; i < num_graphs; i++) {
    graphs[i]->num_updates += cudaUpdateParams[i]->num_inserted_updates;
  }
  
  auto flush_end = std::chrono::steady_clock::now();

  std::cout << "  Flushed Ended.\n";

  std::cout << "Update Kernel finished.\n";

  // End timer for kernel
  auto ins_end = std::chrono::steady_clock::now();

  std::cout << "Number of inserted updates for each subgraph:\n";
  int num_zero_graphs = 0;
  for (int i = 0; i < num_graphs; i++) {
    std::cout << "  Subgraph G_" << i << ": " << graphs[i]->num_updates << "\n";
    if (graphs[i]->num_updates == 0) num_zero_graphs++;
  }
  
  std::cout << "Getting k = " << k << " spanning forests\n";

  std::chrono::duration<double> spanning_forests_time = std::chrono::nanoseconds::zero();
  std::chrono::duration<double> cert_write_time = std::chrono::nanoseconds::zero();
  std::chrono::duration<double> viecut_time = std::chrono::nanoseconds::zero();

  // Get spanning forests then create a METIS format file
  std::cout << "Generating Certificates...\n";
  int num_sampled_zero_graphs = 0;
  for (int i = 0; i < num_graphs - num_zero_graphs; i++) {
    std::cout << "Subgraph G_" << i << ":\n";

    auto spanning_forests_start = std::chrono::steady_clock::now();
    std::vector<std::vector<Edge>> forests = graphs[i]->k_spanning_forests(k, i);
    spanning_forests_time += std::chrono::steady_clock::now() - spanning_forests_start;

    int sampled_edges = 0;
    for (int k_id = 0; k_id < k; k_id++) {
      sampled_edges += forests[k_id].size();
    }
    std::cout << "  Total sampled edges: " << sampled_edges << "\n";

    if(sampled_edges == 0) {
      num_sampled_zero_graphs++;
      continue;
    }

    auto cert_write_start = std::chrono::steady_clock::now();
    
    std::string file_name = "certificates" + std::to_string(i) + ".metis";
    std::ofstream cert (file_name);

    // Read edges then categorize them based on src node
    int sampled_num_nodes = 0;
    int sampled_num_edges = 0;
    node_id_t current_node_id = 1;
    int num_self_edges = 0;

    std::map<node_id_t, std::vector<node_id_t>> nodes_list;
    std::map<node_id_t, node_id_t> simplified_node_ids;

    for (auto forest : forests) {
      for (auto e : forest) {
        if (simplified_node_ids.find(e.src) == simplified_node_ids.end()) { // Has not been inserted yet
          simplified_node_ids[e.src] = current_node_id;
          nodes_list[current_node_id] = std::vector<node_id_t>();

          sampled_num_nodes++;
          current_node_id++;
        }

        if (simplified_node_ids.find(e.dst) == simplified_node_ids.end()) {
          simplified_node_ids[e.dst] = current_node_id;
          nodes_list[current_node_id] = std::vector<node_id_t>();

          sampled_num_nodes++;
          current_node_id++;
        }
      
        node_id_t simplified_node1 = simplified_node_ids[e.src];
        node_id_t simplified_node2 = simplified_node_ids[e.dst];
        
        if (simplified_node1 == simplified_node2) {
          num_self_edges++;
        }
        
        nodes_list[simplified_node1].push_back(simplified_node2);
        nodes_list[simplified_node2].push_back(simplified_node1);

        sampled_num_edges++;
      }
    }

    if (num_self_edges > 0) {
      std::cout << "WARNING: There are self edges! " << num_self_edges << "\n";
    }

    // Write sampled num_nodes and num_edges to file
    cert << sampled_num_nodes << " " << sampled_num_edges << " 0" << "\n";

    for (auto it : nodes_list) {
      for (size_t neighbor = 0; neighbor < it.second.size(); neighbor++) {
        if (it.second[neighbor] == it.first) {
          continue;
        }
        cert << (it.second[neighbor]) << " ";
      }
      cert << "\n";  
    }
    cert.close();
    cert_write_time += std::chrono::steady_clock::now() - cert_write_start;
  }


  std::cout << "Getting minimum cut of certificates...\n";
  auto viecut_start = std::chrono::steady_clock::now();
  std::vector<int> mincut_values;
  for (int i = 0; i < num_graphs - num_zero_graphs - num_sampled_zero_graphs; i++) {
    std::string file_name = "certificates" + std::to_string(i) + ".metis";
    std::string output_name = "mincut" + std::to_string(i) + ".txt";
    std::string command = "../VieCut/build/mincut_parallel " + file_name + " exact >" + output_name; // Run VieCut and store the output
    std::system(command.data());

    std::string line;
    std::ifstream output_file(output_name);
    if(output_file.is_open()) {
      std::getline(output_file, line); // Skip first line
      std::getline(output_file, line);

      int start_index = line.find("cut=");
      int end_index = line.find(" n=");

      if (start_index != std::string::npos || end_index != std::string::npos ) {
        int cut = stoi(line.substr((start_index + 4), ((end_index) - (start_index + 4)))); 
        std::cout << "  G_" << i << ": " << cut << "\n";
        mincut_values.push_back(cut);
      }
      else {
        std::cout << "Error: Couldn't find 'cut=' or 'n=' in the output file\n";
      }
      output_file.close();
    }
    else {
      std::cout << "Error: Couldn't find file name: " << output_name << "!\n";
    }
  }
  viecut_time += std::chrono::steady_clock::now() - viecut_start;

  // Go through min cut values of each subgraph and find the minimum cut of subgraph that is smaller than k
  for (int i = 0; i < mincut_values.size(); i++) {
    if(mincut_values[i] < k) {
      std::cout << "Mincut value found! i: " << i << " mincut: " << mincut_values[i] << "\n";
      std::cout << "Final mincut value: " << (mincut_values[i] * (pow(2, i))) << "\n";
      break;
     }
  }

  for (int i = 0; i < num_graphs; i++) {
    delete graphs[i];
  }

  std::chrono::duration<double> insert_time = flush_end - ins_start;
  std::chrono::duration<double> flush_time = flush_end - flush_start;

  double num_seconds = insert_time.count();
  std::cout << "Total insertion time(sec): " << num_seconds << std::endl;
  std::cout << "Updates per second: " << stream.edges() / num_seconds << std::endl;
  std::cout << "Flush Gutters(sec): " << flush_time.count() << std::endl;
  std::cout << "Spanning Forests Time(sec): " << spanning_forests_time.count() << std::endl;

  double total_sampling_forests_time = 0;
  double total_trimming_forests_time = 0;

  for (int i = 0; i < num_graphs; i++) {
    total_sampling_forests_time += graphs[i]->sampling_forests_time.count();
    total_trimming_forests_time += graphs[i]->trimming_forests_time.count();
  }

  std::cout << "  Total Sampling Forests Time(sec): " << total_sampling_forests_time << std::endl;
  std::cout << "  Total Trimming Forests Time(sec): " << total_trimming_forests_time << std::endl;

  std::cout << "Certificate Writing Time(sec): " << cert_write_time.count() << std::endl;
  std::cout << "VieCut Program Time(sec): " << viecut_time.count() << std::endl;

}
